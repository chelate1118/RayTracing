#include <stdio.h>
#include <hip/hip_runtime.h>

typedef unsigned usize;

__global__ void addArrayInt(const int* arr1, const int* arr2, int* sum, usize length) {
    usize i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < length)
        sum[i] = arr1[i] + arr2[i];
}

extern "C" {
    int* add_array_int(const int* arr1, const int* arr2, usize length) {
        size_t size = sizeof(float) * length;

        int* sum = (int*)malloc(size);

        int* copy1 = NULL;
        int* copy2 = NULL;
        int* copy_sum = NULL;

        auto err1 = hipMalloc((void**)&copy1, size);
        auto err2 = hipMalloc((void**)&copy2, size);
        auto err3 = hipMalloc((void**)&copy_sum, size);

        if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
            fprintf(stderr, "Failed to allocate device array");
            exit(EXIT_FAILURE);
        }

        err1 = hipMemcpy(copy1, arr1, size, hipMemcpyHostToDevice);
        err2 = hipMemcpy(copy2, arr2, size, hipMemcpyHostToDevice);

        if (err1 != hipSuccess || err2 != hipSuccess) {
            fprintf(stderr, "Failed to copy host to device");
            exit(EXIT_FAILURE);
        }

        int threadsPerBlock = 1024;
        int blocksPerGrid = (length - 1) / threadsPerBlock + 1;

        addArrayInt<<<threadsPerBlock, blocksPerGrid>>>(copy1, copy2, copy_sum, length);

        err1 = hipGetLastError();

        if (err1 != hipSuccess) {
            fprintf(stderr, "Failed to launch kernel");
            exit(EXIT_FAILURE);
        }

        err1 = hipMemcpy(sum, copy_sum, size, hipMemcpyDeviceToHost);

        if (err1 != hipSuccess) {
            fprintf(stderr, "Failed to copy device to host");
            exit(EXIT_FAILURE);
        }

        err1 = hipFree(copy1);
        err2 = hipFree(copy2);
        err3 = hipFree(copy_sum);

        if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
            fprintf(stderr, "Failed to free device");
            exit(EXIT_FAILURE);
        }

        hipDeviceReset();

        return sum;
    }
}