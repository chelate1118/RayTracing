#include <stdio.h>
#include <hip/hip_runtime.h>

typedef unsigned usize;

__global__ void addArrayInt(const int* arr1, const int* arr2, int* sum, usize length) {
    usize i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < length)
        sum[i] = arr1[i] + arr2[i];
}

__host__ void safeCall(hipError_t err, const char* message) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s (%s)", message, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

template<typename T>
__host__ T* allocCudaMem(size_t size) {    
    T* ptr = NULL;

    safeCall(
        hipMalloc((void**)&ptr, size),
        "Failed to allocate device memory."
    );

    return ptr;
}

template<typename T>
__host__ T* toCudaMem(const T* ptr, size_t size) {
    T* copy_ptr = allocCudaMem<T>(size);

    safeCall(
        hipMemcpy(copy_ptr, ptr, size, hipMemcpyHostToDevice),
        "Failed to copy memory from host to device."
    );

    return copy_ptr;
}

template<typename T>
__host__ T* fromCudaMem(const T* ptr, size_t size) {
    T* host_ptr = (T*)malloc(size);

    safeCall(
        hipMemcpy(host_ptr, ptr, size, hipMemcpyDeviceToHost),
        "Failed to copy memory from device to host."
    );

    return host_ptr;
}

extern "C" {
    int* add_array_int(const int* arr1, const int* arr2, usize length) {
        size_t size = sizeof(float) * length;

        int* copy1 = toCudaMem(arr1, size);
        int* copy2 = toCudaMem(arr2, size);
        int* copy_sum = allocCudaMem<int>(size);

        int blocksPerGrid = 1<<10;
        int threadsPerBlock = (length - 1) / blocksPerGrid + 1;

        addArrayInt<<<threadsPerBlock, blocksPerGrid>>>(copy1, copy2, copy_sum, length);

        safeCall(
            hipGetLastError(),
            "Failed to launch kernel: addArrayInt"
        );

        int* sum = fromCudaMem(copy_sum, size);

        safeCall(hipFree(copy1), "Failed to free device");
        safeCall(hipFree(copy2), "Failed to free device");
        safeCall(hipFree(copy_sum), "Failed to free device");

        hipDeviceReset();

        return sum;
    }
}